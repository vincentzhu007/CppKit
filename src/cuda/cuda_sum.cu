//
// Created by zgd on 2023/10/2.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <iomanip>

void InitWithRandom(float *f, int n_elem) {
  for (int i = 0; i < n_elem; i++) {
    f[i] = rand() * 1.0 / RAND_MAX;
  }
}

void CpuAdd(int n_elem, float *a, float *b, float *c) {
  for (int i = 0; i < n_elem; i++) {
    c[i] = a[i] + b[i];
  }
}

__global__ void kernelSum(float *d_a, float *d_b, float *d_c) {
  int id = threadIdx.x;
  d_c[id] = d_a[id] + d_b[id];
  printf("  -> cuda kernel id[%d], d_a=%.6f, d_b=%.6f, d_c=%.6f\n", id, d_a[id], d_b[id], d_c[id]);
}

void CudaAdd(int n_elem, float *h_a, float *h_b, float *gpu_ref) {
  float *d_a, *d_b, *d_c;
  int n_size = n_elem * sizeof(float);
  hipMalloc(&d_a, n_size);
  hipMalloc(&d_b, n_size);
  hipMalloc(&d_c, n_size);

  // 拷贝输入到device侧
  hipMemcpy(d_a, h_a, n_size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, n_size, hipMemcpyHostToDevice);

  kernelSum <<<1, n_elem>>>(d_a, d_b, d_c);

  // 拷贝结果到host侧
  hipMemcpy(gpu_ref, d_c, n_size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

bool AllClose(int n_elem, float *h_a, float *h_b, float abs_error = 1e-5) {
  for (int i = 0; i < n_elem; i++) {
    if (h_a[i] - h_b[i] >= abs_error) {
      return false;
    }
  }
  return true;
}

std::string Array2Str(float *a, int n_elem) {
  std::stringstream ss;
  ss << "[";
  for (int i = 0; i < n_elem; i++) {
    ss << std::fixed << std::setprecision(6) << a[i];
    if (i < n_elem - 1) {
      ss << ", ";
    }
  }
  ss << "]";
  return ss.str();
}

int main() {
  printf("CUDA Demo: add two arrays.\n\n");

  int n_elem = 6;

  float *h_a = (float *)malloc(sizeof(float) * n_elem);
  float *h_b = (float *)malloc(sizeof(float) * n_elem);
  float *h_c = (float *)malloc(sizeof(float) * n_elem);

  // Setup inputs.
  InitWithRandom(h_a, n_elem);
  InitWithRandom(h_b, n_elem);

  // Run Add() on CPU.
  CpuAdd(n_elem, h_a, h_b, h_c);

  printf("          a = %s\n", Array2Str(h_a, n_elem).c_str());
  printf("          b = %s\n", Array2Str(h_b, n_elem).c_str());
  printf("CPU:  a + b = %s\n", Array2Str(h_c, n_elem).c_str());

  // Run Add() on CUDA.
  float *gpu_ref = (float *)malloc(sizeof(float) * n_elem);
  CudaAdd(n_elem, h_a, h_b, gpu_ref);
  printf("CUDA: a + b = %s\n", Array2Str(gpu_ref, n_elem).c_str());

  // Compare results.
  printf("CUDA result equals to CPU? %s.",
         AllClose(n_elem, h_c, gpu_ref, 1e-9) ? "True" : "False");

  free(h_a);
  free(h_b);
  free(h_c);
  free(gpu_ref);
}
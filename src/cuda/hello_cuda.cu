
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
  printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

int main(void) {
  printf("Hello World from CPU!\n");

  helloFromGPU <<<1, 5>>>();

  // 销毁当前进程所管理设备的所有资源。
//  cudaDeviceReset();
  hipDeviceSynchronize();

  return 0;
}

#include "hip/hip_runtime.h"
//
// Created by zgd on 2023/10/2.
//
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <iomanip>
#include <sys/time.h>

#define CHECK(call)       \
  do {                    \
    const hipError_t error = call; \
    if (error != hipSuccess) {     \
      printf("Error: %s: %d, ", __FILE__, __LINE__); \
      printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
      exit(1);            \
    }                     \
  } while(0)

double CpuSeconds() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

void InitWithRandom(float *f, int n_elem) {
  for (int i = 0; i < n_elem; i++) {
    f[i] = rand() * 1.0 / RAND_MAX;
  }
}

void CpuAdd(int n_elem, float *a, float *b, float *c) {
  auto start_time = CpuSeconds();
  for (int i = 0; i < n_elem; i++) {
    c[i] = a[i] + b[i];
  }
  printf("cpu add elapsed:  %.6lfs\n", CpuSeconds() - start_time);
}

__global__ void kernelAdd(float *d_a, float *d_b, float *d_c, int n_elem) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n_elem) {
    d_c[id] = d_a[id] + d_b[id];
  }
}

void CudaAdd(int n_elem, float *h_a, float *h_b, float *gpu_ref, int max_thread_per_block) {
  float *d_a, *d_b, *d_c;
  int n_size = n_elem * sizeof(float);
  CHECK(hipMalloc(&d_a, n_size));
  CHECK(hipMalloc(&d_b, n_size));
  CHECK(hipMalloc(&d_c, n_size));

  // 拷贝输入到device侧
  CHECK(hipMemcpy(d_a, h_a, n_size, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b, h_b, n_size, hipMemcpyHostToDevice));


  dim3 block(std::min(n_elem, max_thread_per_block));
  dim3 grid((n_elem + block.x - 1) / block.x);
  printf("cuda block:(%d, %d, %d), grid:(%d, %d, %d)\n",
         block.x, block.y, block.z, grid.x, grid.y, grid.z);

  auto start_time = CpuSeconds();
  // kernel调用是异步的，下句执行后会立即返回到host侧，需要手动调用cuda同步函数等待所有线程执行完毕。
  kernelAdd<<<grid, block>>>(d_a, d_b, d_c, n_elem);
  hipDeviceSynchronize();
  printf("cuda add elapsed: %.6lfs\n", CpuSeconds() - start_time);

  // 拷贝结果到host侧
  CHECK(hipMemcpy(gpu_ref, d_c, n_size, hipMemcpyDeviceToHost));

  CHECK(hipFree(d_a));
  CHECK(hipFree(d_b));
  CHECK(hipFree(d_c));
}

bool AllClose(int n_elem, float *cpu_data, float *gpu_data, float abs_error = 1e-5) {
  for (int i = 0; i < n_elem; i++) {
    if (cpu_data[i] - gpu_data[i] >= abs_error) {
      printf("cpu: %f and gpu: %f mismatched at index %d.\n",
             cpu_data[i], gpu_data[i], i);
      return false;
    }
  }
  return true;
}

std::string Array2Str(float *a, int n_elem, int visible_elem = 10) {
  std::stringstream ss;
  ss << "[";
  for (int i = 0; i < n_elem && i < visible_elem; i++) {
    ss << std::fixed << std::setprecision(6) << a[i];
    if (i < n_elem - 1) {
      ss << ", ";
    }
  }
  if (visible_elem < n_elem) {
    ss << "...";
  }
  ss << "]";
  return ss.str();
}

int main() {
  printf("CUDA Demo: add two vectors.\n");
  hipDeviceProp_t device_prop;
  int dev = 0;
  CHECK(cudaGetDeviceProperties_v2(&device_prop, dev));
  printf("Using device %d: %s, maxThreadsPerBlock: %d.\n\n",
         dev, device_prop.name, device_prop.maxThreadsPerBlock);
  CHECK(hipSetDevice(dev));

  std::vector<int> elem_levels = {1<<24};
  for (auto n_elem: elem_levels) {
    printf("testing n_elem: %d ...\n", n_elem);
    float *h_a = (float *)malloc(sizeof(float) * n_elem);
    float *h_b = (float *)malloc(sizeof(float) * n_elem);
    float *h_c = (float *)malloc(sizeof(float) * n_elem);

    // Fill inputs with random data.
    InitWithRandom(h_a, n_elem);
    InitWithRandom(h_b, n_elem);

    // Run Add() on CPU.
    CpuAdd(n_elem, h_a, h_b, h_c);

    // Run Add() on CUDA.
    float *gpu_ref = (float *)malloc(sizeof(float) * n_elem);
    int max_thread_per_block = device_prop.maxThreadsPerBlock;
    CudaAdd(n_elem, h_a, h_b, gpu_ref, max_thread_per_block);
    printf("cpu result : %s\n", Array2Str(h_c, n_elem).c_str());
    printf("cuda result: %s\n", Array2Str(gpu_ref, n_elem).c_str());

    // Compare results.
    printf("cuda equals to cpu? %s.\n\n",
           AllClose(n_elem, h_c, gpu_ref, 1e-9) ? "yes" : "no");

    free(h_a);
    free(h_b);
    free(h_c);
    free(gpu_ref);
  }
  return 0;
}
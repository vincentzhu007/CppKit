//
// Created by zgd on 2023/10/2.
//

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void CheckIndex(void) {
  printf("Device: threadIdx:(%d, %d, %d), blockIdx:(%d, %d, %d), "
         "blockDim:(%d, %d, %d), gridDim:(%d, %d, %d)\n",
         threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z,
         blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main() {
  printf("CUDA Demo: illustrate block and thread index.\n\n");

  int n_elem = 6;
  // 计算block和grid数量：1个grid包含多个block，1个block包含多个thread.
  dim3 block(3); // block内的thread分布
  dim3 grid((n_elem + block.x - 1) / block.x); // grid中的block分布

  printf("Host: block:(%d, %d, %d), grid:(%d, %d, %d)\n\n",
         block.x, block.y, block.z, grid.x, grid.y, grid.z);

  CheckIndex<<<grid, block>>>();
  hipDeviceReset();
}